#include <stdio.h>
#include <hip/hip_runtime.h>

#include <linux/kernel.h>
#include <time.h>

clock_t start, end;
double cpu_time_used;

#define N 10000000
#define K 16
#define NUM_THREADS_PER_BLOCK 256

float *points_x;
float *points_y;
int *cluster;

float *centroid_x;
float *centroid_y;

float *newCentroid_x;
float *newCentroid_y;

int *size;

int iterations = 0;

float *d_centroid_x, *d_centroid_y, *d_points_x, *d_points_y;
int *d_cluster;

// __device__ static inline float distance(float p1_x, float p1_y, float p2_x, float p2_y) {
//     //No need for the actual value of the distance. We can avoid the sqrt() function since we'll only be comparing which one is the shortest distance.
//     return ((p2_x - p1_x)*(p2_x - p1_x)) + ((p2_y - p1_y)*(p2_y - p1_y));
// }

void initialize(){
    points_x = (float *)malloc(N * sizeof(float));
    points_y = (float *)malloc(N * sizeof(float));
    cluster = (int *)malloc(N * sizeof(int));

    centroid_x = (float *)malloc(K * sizeof(float));
    centroid_y = (float *)malloc(K * sizeof(float));

    newCentroid_x = (float *)malloc(K * sizeof(float));
    newCentroid_y = (float *)malloc(K * sizeof(float));
    size = (int *)malloc(K * sizeof(int));

    hipMalloc(&d_centroid_x, K*sizeof(float));
    hipMalloc(&d_centroid_y, K*sizeof(float));
    hipMalloc(&d_points_x, N*sizeof(float));
    hipMalloc(&d_points_y, N*sizeof(float));
    hipMalloc(&d_cluster, N*sizeof(int));

    srand(10);
// Random values for the points
    for (int i = 0; i < N; i++){
	points_x[i] = (float)rand() / RAND_MAX;
	points_y[i] = (float)rand() / RAND_MAX;
    }
    for (int i = 0; i < K; i++){
	centroid_x[i] = points_x[i];
	centroid_y[i] = points_y[i];
    }

    hipMemcpy(d_centroid_x, centroid_x, K*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_centroid_y, centroid_y, K*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_points_x, points_x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_points_y, points_y, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_cluster, cluster, N*sizeof(int), hipMemcpyHostToDevice);
}

__global__ void kernel_computeDistances (float *d_centroid_x, float *d_centroid_y, float *d_points_x, float *d_points_y, int *d_cluster) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
	float smallerDist = ((d_points_x[i] - d_centroid_x[0]) * (d_points_x[i] - d_centroid_x[0])) + ((d_points_y[i] - d_centroid_y[0]) * (d_points_y[i] - d_centroid_y[0]));
	int clusterIndex = 0;
	for (int j = 1; j < K; j++){
	    float tempDist = ((d_points_x[i] - d_centroid_x[j]) * (d_points_x[i] - d_centroid_x[j])) + ((d_points_y[i] - d_centroid_y[j]) * (d_points_y[i] - d_centroid_y[j]));
	    clusterIndex = tempDist < smallerDist ? j : clusterIndex;
	    smallerDist = tempDist < smallerDist ? tempDist : smallerDist;
	}
	//Stores cluster index associated to each point
	d_cluster[i] = clusterIndex;
    }
}

void k_means(){
    for(iterations = 0; iterations < 20; iterations++) {
	// Each iteration resets every cluster
	for (int i = 0; i < K; i++){
	    size[i] = 0;
	    newCentroid_x[i] = 0.0;
	    newCentroid_y[i] = 0.0;
	}
    int blocks = (N + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;
	kernel_computeDistances<<<blocks, NUM_THREADS_PER_BLOCK>>>(d_centroid_x, d_centroid_y, d_points_x, d_points_y, d_cluster);
	hipDeviceSynchronize();
	 hipError_t err = hipGetLastError();
	 if (err != hipSuccess)
        {
            printf("1 -> Kernel launch failed: %s\n", hipGetErrorString(err));
            return;
        }
        hipMemcpy(centroid_x, d_centroid_x, K*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(centroid_y, d_centroid_y, K*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(points_x, d_points_x, N*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(points_y, d_points_y, N*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(cluster, d_cluster, N*sizeof(int), hipMemcpyDeviceToHost);

        for (int i = 0; i < N; i++) {
            // Sums the value of the point to the newCentroid variable.
            size[cluster[i]]++;
            newCentroid_x[cluster[i]] += points_x[i];
            newCentroid_y[cluster[i]] += points_y[i];
        }

    // Checks if the newCentroid is the same as the current centroid (stopping case).
    // Sets the current centroid to the value of the newCentroid.
    for (int i = 0; i < K; i++){
        newCentroid_x[i] /= size[i];
        newCentroid_y[i] /= size[i];
        centroid_x[i] = newCentroid_x[i];
        centroid_y[i] = newCentroid_y[i];
    }

    hipMemcpy(d_centroid_x, centroid_x, K*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_centroid_y, centroid_y, K*sizeof(float), hipMemcpyHostToDevice);

    }
}

static inline void printEndMessage(){
    printf("\nN: %d, K = %d\n", N, K);
    for (int i = 0; i < K; i++){
        printf("Center: (%.3f, %.3f) : size: %d\n", centroid_x[i], centroid_y[i], size[i]);
    }
    printf("Iterations: %d\n", iterations);
}

int main(int argc, char *argv[]){
    start = clock();
    initialize();
    k_means();
    end = clock();
    printEndMessage();
    printf("Work took %f seconds\n", end - start / CLOCKS_PER_SEC);
    return 0;
}